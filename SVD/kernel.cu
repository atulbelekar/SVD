#include "hip/hip_runtime.h"
﻿#include <hip/device_functions.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<iostream>
#include "Matrix.h"
#include "hip/device_functions.h"
#include<random>
using namespace std;


__global__ void matmul(Matrix a, Matrix b, Matrix c) {
    
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < a.row; i+= (blockDim.x * gridDim.x)) {
        for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < b.col; j+=(blockDim.x * gridDim.x)) {
            double sum = 0;
            for (int k = 0; k < a.row; k++) {
                sum += (a.dp[i * a.col + k] * b.dp[k * b.col + j]);
            }
            c.dp[i * c.col + j] = sum;
        }
    }
}

__global__ void Gram_schmidt(Matrix a, Matrix ans) {
    
    for (int i = 0; i < a.col; i++) {
        for (int j = i - 1; j >= 0; j--) {
            double dot = dot_prod(a, ans, i, j);

            for (int k = 0; k < a.row; k++) {
                ans.dp[k*(ans.col)+i] += dot* (ans.dp[k*(ans.col)+j]);
            }
        }
        for (int k = 0; k < a.row; k++) {
            ans.dp[k * (ans.col) + i] = (ans.dp[k * (ans.col) + i] - a.dp[k * (ans.col) + i]);
        }
        double norm_temp = norm(ans,i);
        for (int k = 0; k < a.row; k++) {
            ans.dp[k * (ans.col) + i] = ans.dp[k * (ans.col) +i] / norm_temp;
        }
    }
    
}

__global__ void R_matrix(Matrix a, Matrix Q, Matrix R) {

    for (int i = 0; i < a.row; i++) {
        for (int j = i; j < a.col; j++) {
            R.dp[i*(R.col)+j] = dot_prod(a, Q, j, i);
        }
    }
}


int main() {
   
    Matrix A(2, 2);
    A.random_init(0, 10);
    A.cuda_malloc();
    dim3 size(10, 10);
    int n = 20;
    Matrix Q(2, 2);
    Q.cuda_malloc();
    Matrix R(2, 2);
    R.cuda_malloc();
    while (n--) {
        cout << "Q : \n";
        Q.cuda_copy_to_host();
        Q.print();
        cout << "R: \n";
        R.cuda_copy_to_host();
        R.print();
        cout << "A: \n";
        A.cuda_copy_to_host();
        A.print();
        cout << "****\n";
        Q.set_zero();
        Q.cuda_free();
        Q.set_zero();
        Q.cuda_malloc();
        Gram_schmidt << <1, size >> > (A, Q);
        R_matrix << <1, size >> > (A, Q, R);
        matmul << <1, size >> > (R, Q, A);
        
    }
    


}

