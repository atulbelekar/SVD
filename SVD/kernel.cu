#include "hip/hip_runtime.h"
﻿#include <hip/device_functions.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<iostream>
#include "Matrix.h"
#include "helper_functions.h"
#include<random>
using namespace std;

__global__ void matmul(Matrix a, Matrix b, Matrix c) {
    
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < a.row; i+= (blockDim.x * gridDim.x)) {
        for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < a.col; j += (blockDim.y * gridDim.x)){
            double sum = 0;
            for (int k = 0; k < a.row; k++) {
                sum += (a.dp[i * a.col + k] * b.dp[k * b.col + j]);
            }
            c.dp[i * c.col + j] = sum;
        }
    }
}

__global__ void Gram_schmidt(Matrix a, Matrix ans) {
    
    for (int i =0; i < a.row; i++) {
        for (int j = i - 1; j >= 0; j--) {
            double dot = dot_prod(a, ans, i, j);

            for (int k = 0; k < a.row; k++) {
                ans.dp[k*(ans.col)+i] += dot* (ans.dp[k*(ans.col)+j]);
            }
        }
        for (int k = 0; k < a.row; k++) {
            ans.dp[k * (ans.col) + i] = (ans.dp[k * (ans.col) + i] - a.dp[k * (ans.col) + i]);
        }
        double norm_temp = norm(ans,i);
        for (int k = 0; k < a.row; k++) {
            ans.dp[k * (ans.col) + i] = ans.dp[k * (ans.col) +i] / norm_temp;
        }
    }
    
}

__global__ void R_matrix(Matrix a, Matrix Q, Matrix R) {

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < a.row; i += (blockDim.x * gridDim.x)) {
        for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < a.col; j += (blockDim.y * gridDim.x)) {
            R.dp[i*(R.col)+j] = dot_prod(a, Q, j, i);
        }
    }
}

void Find_eigen_values(Matrix &A, Matrix &Q, int n) {
    Matrix R(A.row, A.col);
    R.cuda_malloc();
    dim3 size(16, 16);
    while (n--) {
        Q.cuda_free();
        Q.set_zero();
        Q.cuda_malloc();
        Gram_schmidt << <1, 1 >> > (A, Q);
        R_matrix << <1, size >> > (A, Q, R);
        matmul << <1, size >> > (R, Q, A);
    }
}

Matrix InverseOfMatrix(Matrix A)
{
    Matrix I(A.row,A.col);
    I.make_identity();

    for (int i = 0; i < A.row-1; i++) {
        for (int j = i + 1; j < A.row; j++) {
            double temp=A.p[j*A.col+i]/A.p[i*A.col+i];
            for (int k = 0; k < A.row; k++) {
                A.p[j * A.row + k] = A.p[j * A.row + k] - A.p[i * A.row + k] * temp;
                I.p[j * A.row + k] = I.p[j * A.row + k] - I.p[i * A.row + k] * temp;
            }
        }
    }
    for (int i = A.row - 1; i >0; i--) {
        for (int j = i - 1; j >= 0; j--) {
            double temp = A.p[j * A.col + i] / A.p[i * A.col + i];
            for (int k = 0; k < A.row; k++) {
                A.p[j * A.row + k] = A.p[j * A.row + k] - A.p[i * A.row + k] * temp;
                I.p[j * A.row + k] = I.p[j * A.row + k] - I.p[i * A.row + k] * temp;
            }
        }
    }
    for (int i = 0; i < A.row; i++) {
        for (int j = 0; j < A.row; j++) {
            I.p[i * A.row + j] = I.p[i * A.row + j] / A.p[i * A.row + i];
        }
    }
    return I;
}

int main() {
   
    
}

