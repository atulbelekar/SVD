#include "hip/hip_runtime.h"
﻿#include <hip/device_functions.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<iostream>
#include "Matrix.h"
#include<random>
using namespace std;


__global__ void matmul(Matrix a, Matrix b, Matrix c) {
    
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < a.row; i+= (blockDim.x * gridDim.x)) {
        for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < b.col; j+=(blockDim.x * gridDim.x)) {
            double sum = 0;
            for (int k = 0; k < a.row; k++) {
                sum += (a.dp[i * a.col + k] * b.dp[k * b.col + j]);
            }
            c.dp[i * c.col + j] = sum;
        }
    }
}


int main() {
    
    
    
}

